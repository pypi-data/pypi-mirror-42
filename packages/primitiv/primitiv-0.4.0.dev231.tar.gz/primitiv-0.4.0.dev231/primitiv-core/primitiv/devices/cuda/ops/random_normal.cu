#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/devices/cuda/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace primitiv {
namespace devices {

void CUDA::random_normal_impl(float mean, float sd, Tensor &y) {
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateNormal(
        state_->hiprand.get(), MDATA(y), y.shape().size(), mean, sd));
}

}  // namespace devices
}  // namespace primitiv
