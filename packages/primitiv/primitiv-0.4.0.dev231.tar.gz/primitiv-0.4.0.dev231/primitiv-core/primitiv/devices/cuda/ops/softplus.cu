#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/devices/cuda/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

CUDADEV_KERNEL_FW_X(
    softplus, ::fmaxf(px[i], .0f) + ::logf(1.f + ::expf(-::fabs(px[i]))));
CUDADEV_KERNEL_BW_X(softplus, (.5f + .5f * ::tanhf(.5f * px[i])) * pgy[i]);

}  // namespace

namespace primitiv {
namespace devices {

CUDADEV_FW_X(softplus);
CUDADEV_BW_X(softplus);

}  // namespace devices
}  // namespace primitiv
