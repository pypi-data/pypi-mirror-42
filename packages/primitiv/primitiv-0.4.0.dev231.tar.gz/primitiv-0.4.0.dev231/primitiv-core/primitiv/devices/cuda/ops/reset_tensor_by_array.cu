#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/internal/cuda/utils.h>
#include <primitiv/devices/cuda/ops/common.h>

namespace primitiv {
namespace devices {

void CUDA::reset_tensor_by_array_impl(const float values[], Tensor &x) {
  const std::uint32_t size = x.shape().size();
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        MDATA(x), values, sizeof(float) * size, hipMemcpyHostToDevice));
}

}  // namespace devices
}  // namespace primitiv
