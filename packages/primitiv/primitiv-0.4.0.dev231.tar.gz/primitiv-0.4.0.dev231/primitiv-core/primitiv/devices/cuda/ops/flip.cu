#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/internal/cuda/utils.h>
#include <primitiv/devices/cuda/ops/common.h>

namespace {

__global__ void flip_fw_dev(
    const float *px, std::uint32_t skip, std::uint32_t n, std::uint32_t r, float *py) {
  const unsigned i = IDX;
  const unsigned j = IDY;
  const unsigned offset = j * n - j % skip * (n - 1);
  if (i < n && j < r) {
    py[offset + i * skip] = px[offset + (n - i - 1) * skip];
  }
}

__global__ void flip_bw_dev(
    const float *pgy, std::uint32_t skip, std::uint32_t n, std::uint32_t r, float *pgx) {
  const unsigned i = IDX;
  const unsigned j = IDY;
  const unsigned offset = j * n - j % skip * (n - 1);
  if (i < n && j < r) {
    pgx[offset + i * skip] += pgy[offset + (n - i - 1) * skip];
  }
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA::flip_fw_impl(
    const Tensor &x, std::uint32_t dim, Tensor &y) {
  const Shape &s = x.shape();
  const std::uint32_t n = s[dim];
  const std::uint32_t skip = s.lower_volume(dim);
  const std::uint32_t r = s.size() / n;
  const std::uint32_t g1 = GRID_SIZE(n, dim2_x_);
  const std::uint32_t g2 = GRID_SIZE(r, dim2_y_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::flip_fw_dev<<<dim3(g1, g2, 1), dim3(dim2_x_, dim2_y_, 1)>>>(
      CDATA(x), skip, n, r, MDATA(y));
}

void CUDA::flip_bw_impl(
    const Tensor &gy, std::uint32_t dim, Tensor &gx) {
  const Shape &s = gy.shape();
  const std::uint32_t n = s[dim];
  const std::uint32_t skip = s.lower_volume(dim);
  const std::uint32_t r = s.size() / n;
  const std::uint32_t g1 = GRID_SIZE(n, dim2_x_);
  const std::uint32_t g2 = GRID_SIZE(r, dim2_y_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::flip_bw_dev<<<dim3(g1, g2, 1), dim3(dim2_x_, dim2_y_, 1)>>>(
      CDATA(gy), skip, n, r, MDATA(gx));
}

}  // namespace devices
}  // namespace primitiv
