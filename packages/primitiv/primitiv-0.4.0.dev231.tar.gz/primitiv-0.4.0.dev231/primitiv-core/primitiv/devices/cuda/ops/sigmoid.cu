#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/devices/cuda/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

CUDADEV_KERNEL_FW_X(sigmoid, .5f + .5f * ::tanhf(.5f * px[i]));
CUDADEV_KERNEL_BW_X(sigmoid, py[i] * (1.f - py[i]) * pgy[i]);

}  // namespace

namespace primitiv {
namespace devices {

CUDADEV_FW_X(sigmoid);
CUDADEV_BW_X(sigmoid);

}  // namespace devices
}  // namespace primitiv
