#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/devices/cuda/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

CUDADEV_KERNEL_FW_X_CONST(multiply_const, px[i] * k);
CUDADEV_KERNEL_BW_X_CONST(multiply_const, k * pgy[i]);

CUDADEV_KERNEL_FW_X_SCALAR_R(multiply_scalar, ::__fmul_rn);

CUDADEV_KERNEL_FW_AB(multiply, ::__fmul_rn);

__global__ void multiply_bw_dev(
    const float *pa, const float *pb, const float *, const float *pgy,
    std::uint32_t size, std::uint32_t mba, std::uint32_t mbb,
    float *pga, float *pgb) {
  const std::uint32_t i = IDX;
  const std::uint32_t shift = blockIdx.y * size;
  if (i < size) {
    const float gy = pgy[i + shift];
    const std::uint32_t a_ofs = i + mba * shift;
    const std::uint32_t b_ofs = i + mbb * shift;
    ::atomicAdd(pga + a_ofs, gy * pb[b_ofs]);
    ::atomicAdd(pgb + b_ofs, gy * pa[a_ofs]);
  }
}

}  // namespace

namespace primitiv {
namespace devices {

CUDADEV_FW_X_CONST(multiply_const);
CUDADEV_BW_X_CONST(multiply_const);

CUDADEV_FW_X_SCALAR(multiply_scalar);

CUDADEV_FW_AB(multiply);
CUDADEV_BW_AB(multiply);

}  // namespace devices
}  // namespace primitiv
