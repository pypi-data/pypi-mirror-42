#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/devices/cuda/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

CUDADEV_KERNEL_FW_X(sqrt, ::__fsqrt_rn(px[i]));
CUDADEV_KERNEL_BW_X(sqrt, .5f * pgy[i] / py[i]);

}  // namespace

namespace primitiv {
namespace devices {

CUDADEV_FW_X(sqrt);
CUDADEV_BW_X(sqrt);

}  // namespace devices
}  // namespace primitiv
