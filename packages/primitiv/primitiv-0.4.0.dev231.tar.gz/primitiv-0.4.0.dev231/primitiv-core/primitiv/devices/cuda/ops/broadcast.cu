#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/devices/cuda/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void broadcast_fw_dev(
    const float *px,
    std::uint32_t skip1, std::uint32_t skip2, std::uint32_t size,
    float *py) {
  const std::uint32_t i = IDX;
  if (i < size) py[i] = px[i % skip1 + (i / skip2) * skip1];
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA::broadcast_fw_impl(
    const Tensor &x, std::uint32_t dim, std::uint32_t size, Tensor &y) {
  const std::uint32_t skip1 = y.shape().lower_volume(dim);
  const std::uint32_t skip2 = skip1 * size;
  const std::uint32_t total = y.shape().size();
  const std::uint32_t g1 = GRID_SIZE(total, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::broadcast_fw_dev<<<g1, dim1_x_>>>(CDATA(x), skip1, skip2, total, MDATA(y));
}

}  // namespace devices
}  // namespace primitiv
