#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/devices/cuda/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void set_gemm_ptrs(
    const float *pa, const float *pb, const float *py,
    std::uint32_t na, std::uint32_t nb, std::uint32_t ny, std::uint32_t bs,
    const float **ptrs) {
  const std::uint32_t i = IDX;
  if (i < bs) {
    ptrs[i] = pa + i * na;
    ptrs[i + bs] = pb + i * nb;
    ptrs[i + 2 * bs] = py + i * ny;
  }
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA::matmul_fw_impl(const Tensor &a, const Tensor &b, Tensor &y) {
  const std::uint32_t di = a.shape()[0];
  const std::uint32_t dj = a.shape()[1];
  const std::uint32_t dk = b.shape()[1];
  float alpha = 1.;
  float beta = 0.;

  CUDA_CALL(::hipSetDevice(dev_id_));

  if (a.shape().has_batch()) {
    // Do gemm multiple times.
    const float *pa = CDATA(a);
    const float *pb = CDATA(b);
    float *py = MDATA(y);
    const std::uint32_t na = di * dj;
    const std::uint32_t nb = b.shape().has_batch() * dj * dk;
    const std::uint32_t ny = di * dk;
    const std::uint32_t bs = a.shape().batch();

    std::shared_ptr<void> ptrs = state_->pool.allocate(3 * bs * sizeof(void *));
    const float **fptrs = static_cast<const float **>(ptrs.get());

    const std::uint32_t gs = GRID_SIZE(bs, dim1_x_);

    ::set_gemm_ptrs<<<gs, dim1_x_>>>(pa, pb, py, na, nb, ny, bs, fptrs);
    CUBLAS_CALL(::hipblasSgemmBatched(
          state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
          di, dk, dj,
          &alpha, fptrs, di, fptrs + bs, dj,
          &beta, const_cast<float **>(fptrs) + 2 * bs, di,
          bs));
  } else {
    // Do gemm only once to calculate the product with a combined matrix.
    CUBLAS_CALL(::hipblasSgemm(
          state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
          di, dk * b.shape().batch(), dj,
          &alpha, CDATA(a), di, CDATA(b), dj,
          &beta, MDATA(y), di));
  }
}

void CUDA::matmul_bw_impl(
    const Tensor &a, const Tensor &b, const Tensor &, const Tensor &gy,
    Tensor &ga, Tensor &gb) {
  // ga += gy . b^T
  // gb += a^T . gy
  const std::uint32_t di = a.shape()[0];
  const std::uint32_t dj = a.shape()[1];
  const std::uint32_t dk = b.shape()[1];
  float alpha = 1.;
  float beta = 1.;

  CUDA_CALL(::hipSetDevice(dev_id_));

  if (a.shape().has_batch()) {
    // Do gemm multiple times.
    const float *pa = CDATA(a);
    const float *pb = CDATA(b);
    const float *pgy = CDATA(gy);
    float *pga = MDATA(ga);
    float *pgb = MDATA(gb);
    const std::uint32_t na = di * dj;
    const std::uint32_t nb = b.shape().has_batch() * dj * dk;
    const std::uint32_t ny = di * dk;
    const std::uint32_t bs = a.shape().batch();

    std::shared_ptr<void> ptrs = state_->pool.allocate(3 * bs * sizeof(void *));
    const float **fptrs = static_cast<const float **>(ptrs.get());

    const std::uint32_t gs = GRID_SIZE(bs, dim1_x_);

    ::set_gemm_ptrs<<<gs, dim1_x_>>>(pgy, pb, pga, ny, nb, na, bs, fptrs);
    CUBLAS_CALL(::hipblasSgemmBatched(
          state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
          di, dj, dk,
          &alpha, fptrs, di, fptrs + bs, dj,
          &beta, const_cast<float **>(fptrs) + 2 * bs, di,
          bs));

    if (nb > 0 /* `b` has minibatch */) {
      ::set_gemm_ptrs<<<gs, dim1_x_>>>(pa, pgy, pgb, na, ny, nb, bs, fptrs);
      CUBLAS_CALL(::hipblasSgemmBatched(
            state_->cublas.get(), ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
            dj, dk, di,
            &alpha, fptrs, di, fptrs + bs, di,
            &beta, const_cast<float **>(fptrs) + 2 * bs, dj,
            bs));
    } else {
      // NOTE(odashi):
      // `hipblasSgemmBatched` can not be used due to a data race against
      // shared values in `b` by multiple GEMM operations.
      for (std::uint32_t batch = 0; batch < bs; ++batch) {
        CUBLAS_CALL(::hipblasSgemm(
              state_->cublas.get(), ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
              dj, dk, di,
              &alpha, pa + batch * na, di, pgy + batch * ny, di,
              &beta, pgb, dj));
      }
    }
  } else {
    // Do gemm only once to calculate the product with a combined matrix.
    CUBLAS_CALL(::hipblasSgemm(
          state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
          di, dj, dk * b.shape().batch(),
          &alpha, CDATA(gy), di, CDATA(b), dj,
          &beta, MDATA(ga), di));
    CUBLAS_CALL(::hipblasSgemm(
          state_->cublas.get(), ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
          dj, dk * b.shape().batch(), di,
          &alpha, CDATA(a), di, CDATA(gy), di,
          &beta, MDATA(gb), dj));
  }
}

}  // namespace devices
}  // namespace primitiv
