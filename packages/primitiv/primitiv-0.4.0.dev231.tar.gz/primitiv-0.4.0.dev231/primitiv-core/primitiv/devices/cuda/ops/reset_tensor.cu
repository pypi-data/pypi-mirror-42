#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/internal/cuda/utils.h>
#include <primitiv/devices/cuda/ops/common.h>

namespace {

__global__ void set_const_dev(float k, std::uint32_t size, float *py) {
  const std::uint32_t i = IDX;
  if (i < size) py[i] = k;
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA::reset_tensor_impl(float k, Tensor &x) {
  const std::uint32_t size = x.shape().size();
  const std::uint32_t num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::set_const_dev<<<num_blocks, dim1_x_>>>(k, size, MDATA(x));
}

}  // namespace devices
}  // namespace primitiv
