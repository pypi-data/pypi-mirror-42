#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda/device.h>
#include <primitiv/devices/cuda/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void pick_fw_dev(
    const float *px, const std::uint32_t *pi,
    std::uint32_t wx, std::uint32_t wy,
    std::uint32_t sx, std::uint32_t si, std::uint32_t sy,
    float *py) {
  const std::uint32_t t = IDX;
  const std::uint32_t ox = blockIdx.y * sx + pi[blockIdx.y * si] * wy;
  const std::uint32_t oy = blockIdx.y * sy;
  if (t < sy) py[oy + t] = px[ox + (t / wy) * wx + (t % wy)];
}

__global__ void pick_bw_dev(
    const float *pgy, const std::uint32_t *pi,
    std::uint32_t wx, std::uint32_t wy,
    std::uint32_t sx, std::uint32_t si, std::uint32_t sy,
    float *pgx) {
  const std::uint32_t t = IDX;
  const std::uint32_t ox = blockIdx.y * sx + pi[blockIdx.y * si] * wy;
  const std::uint32_t oy = blockIdx.y * sy;
  if (t < sy) ::atomicAdd(pgx + ox + (t / wy) * wx + (t % wy), pgy[oy + t]);
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA::pick_fw_impl(
    const Tensor &x, const std::vector<std::uint32_t> &ids, std::uint32_t dim,
    Tensor &y) {
  const std::uint32_t wy = y.shape().lower_volume(dim);
  const std::uint32_t sy = y.shape().volume();
  const std::uint32_t g1 = GRID_SIZE(sy, dim1_x_);
  const std::uint32_t bs = y.shape().batch();

  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        ids_ptr_.get(), ids.data(), sizeof(std::uint32_t) * ids.size(),
        hipMemcpyHostToDevice));
  ::pick_fw_dev<<<dim3(g1, bs), dim1_x_>>>(
      CDATA(x), static_cast<const std::uint32_t *>(ids_ptr_.get()),
      wy * x.shape()[dim], wy,
      x.shape().has_batch() * x.shape().volume(), ids.size() > 1, sy,
      MDATA(y));
}

void CUDA::pick_bw_impl(
    const Tensor &gy, const std::vector<std::uint32_t>& ids, std::uint32_t dim,
    Tensor &gx) {
  const std::uint32_t wy = gy.shape().lower_volume(dim);
  const std::uint32_t sy = gy.shape().volume();
  const std::uint32_t g1 = GRID_SIZE(sy, dim1_x_);
  const std::uint32_t bs = gy.shape().batch();

  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        ids_ptr_.get(), ids.data(), sizeof(std::uint32_t) * ids.size(),
        hipMemcpyHostToDevice));
  ::pick_bw_dev<<<dim3(g1, bs), dim1_x_>>>(
      CDATA(gy), static_cast<const std::uint32_t *>(ids_ptr_.get()),
      wy *gx.shape()[dim], wy,
      gx.shape().has_batch() * gx.shape().volume(), ids.size() > 1, sy,
      MDATA(gx));
}

}  // namespace devices
}  // namespace primitiv
