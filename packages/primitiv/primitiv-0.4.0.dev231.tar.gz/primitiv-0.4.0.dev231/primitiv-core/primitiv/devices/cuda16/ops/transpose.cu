#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void transpose_fw_dev(
    const half *px, std::uint32_t rows, std::uint32_t cols, half *py) {
  const std::uint32_t i = IDX;
  const std::uint32_t j = IDY;
  std::uint32_t ofs = blockIdx.z * rows * cols;
  if (i < rows && j < cols) py[ofs + j + i * cols] = px[ofs + i + j * rows];
}

__global__ void transpose_bw_dev(
    const half *py, std::uint32_t rows, std::uint32_t cols, half *px) {
  const std::uint32_t i = IDX;
  const std::uint32_t j = IDY;
  std::uint32_t ofs = blockIdx.z * rows * cols;
  if (i < rows && j < cols) {
    const std::size_t ox = ofs + i + j * rows;
    const std::size_t oy = ofs + j + i * cols;
    INPLACE_ADD(px + ox, ::__half2float(py[oy]));
  }
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::transpose_fw_impl(const Tensor &x, Tensor &y) {
  const std::uint32_t rows = x.shape()[0];
  const std::uint32_t cols = x.shape()[1];
  const std::uint32_t bs = x.shape().batch();
  const std::uint32_t g1 = GRID_SIZE(rows, dim2_x_);
  const std::uint32_t g2 = GRID_SIZE(cols, dim2_y_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::transpose_fw_dev<<<dim3(g1, g2, bs), dim3(dim2_x_, dim2_y_, 1)>>>(
      CDATA(half, x), rows, cols, MDATA(half, y));
}

void CUDA16::transpose_bw_impl(
    const Tensor &, const Tensor &, const Tensor &gy, Tensor &gx) {
  const std::uint32_t rows = gx.shape()[0];
  const std::uint32_t cols = gx.shape()[1];
  const std::uint32_t bs = gx.shape().batch();
  const std::uint32_t g1 = GRID_SIZE(rows, dim2_x_);
  const std::uint32_t g2 = GRID_SIZE(cols, dim2_y_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::transpose_bw_dev<<<dim3(g1, g2, bs), dim3(dim2_x_, dim2_y_, 1)>>>(
      CDATA(half, gy), rows, cols, MDATA(half, gx));
}

}  // namespace devices
}  // namespace primitiv
