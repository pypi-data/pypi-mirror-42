#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void batch_sum_fw_dev(
    const half *px, std::uint32_t size, std::uint32_t batch, half *py) {
  const std::uint32_t i = IDX;
  if (i < size) {
    float temp = .0f;
    px += i;
    for (std::uint32_t j = 0; j < batch; ++j, px += size) {
      temp += ::__half2float(*px);
    }
    py[i] = ::__float2half(temp);
  }
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::batch_sum_fw_impl(const Tensor &x, Tensor &y) {
  const std::uint32_t size = y.shape().size();
  const std::uint32_t g1 = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::batch_sum_fw_dev<<<g1, dim1_x_>>>(
      CDATA(half, x), size, x.shape().batch(), MDATA(half, y));
}

}  // namespace devices
}  // namespace primitiv
