#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

CUDA16_KERNEL_FW_X(
    softplus, ::fmaxf(X_VAL, .0f) + ::logf(1.f + ::expf(-::fabs(X_VAL))));
CUDA16_KERNEL_BW_X(softplus, (.5f + .5f * ::tanhf(.5f * X_VAL)) * GY_VAL);

}  // namespace

namespace primitiv {
namespace devices {

CUDA16_DEV_FW_X(softplus);
CUDA16_DEV_BW_X(softplus);

}  // namespace devices
}  // namespace primitiv
