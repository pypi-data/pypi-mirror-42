#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

CUDA16_KERNEL_FW_X(tanh, ::tanhf(X_VAL));
CUDA16_KERNEL_BW_X(tanh, (1.f - Y_VAL * Y_VAL) * GY_VAL);

}  // namespace

namespace primitiv {
namespace devices {

CUDA16_DEV_FW_X(tanh);
CUDA16_DEV_BW_X(tanh);

}  // namespace devices
}  // namespace primitiv
