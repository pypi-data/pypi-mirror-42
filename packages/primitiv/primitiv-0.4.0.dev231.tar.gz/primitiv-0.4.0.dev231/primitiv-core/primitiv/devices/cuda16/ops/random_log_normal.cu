#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void fp32to16(const float *px, std::size_t size, half *py) {
  const std::size_t i = IDX;
  if (i < size) py[i] = ::__float2half(px[i]);
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::random_log_normal_impl(float mean, float sd, Tensor &y) {
  const std::size_t size = y.shape().size();
  const std::size_t gs = GRID_SIZE(size, dim1_x_);
  auto temp = state_->pool.allocate(size * sizeof(float));
  float *temp_ptr = static_cast<float *>(temp.get());

  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateLogNormal(
        state_->hiprand.get(), temp_ptr, size, mean, sd));
  ::fp32to16<<<gs, dim1_x_>>>(temp_ptr, size, MDATA(half, y));
}

}  // namespace devices
}  // namespace primitiv
