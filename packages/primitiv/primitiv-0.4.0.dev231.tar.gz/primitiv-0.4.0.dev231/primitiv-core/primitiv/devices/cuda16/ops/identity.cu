#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void set_identity_dev(
    std::uint32_t size, std::uint32_t skip, half *py) {
  const std::uint32_t i = IDX;
  if (i < size) py[i] = ::__float2half(!(i % skip));
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::identity_impl(Tensor &y) {
  const std::uint32_t size = y.shape().size();
  const std::uint32_t skip = y.shape()[0] + 1;
  const std::uint32_t num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::set_identity_dev<<<num_blocks, dim1_x_>>>(size, skip, MDATA(half, y));
}

}  // namespace devices
}  // namespace primitiv
