#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void batch_concat_fw_dev(
    const half *px, std::uint32_t y_size, half *py) {
  const std::uint32_t i = IDX;
  if (i < y_size) py[i] = px[i];
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::batch_concat_fw_impl(
    const std::vector<const Tensor *> &xs, Tensor &y) {
  CUDA_CALL(::hipSetDevice(dev_id_));
  std::uint32_t offset = 0;
  for (const Tensor *x : xs) {
    const std::uint32_t span = x->shape().size();
    const std::uint32_t num_blocks = GRID_SIZE(span, dim1_x_);
    ::batch_concat_fw_dev<<<num_blocks, dim1_x_>>>(
       CDATA(half, *x), span, MDATA(half, y) + offset);
    offset += span;
  }
}

}  // namespace devices
}  // namespace primitiv
