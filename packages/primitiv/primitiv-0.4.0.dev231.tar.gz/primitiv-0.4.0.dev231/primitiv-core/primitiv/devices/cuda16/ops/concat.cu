#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void concat_fw_dev(
    const half *px,
    std::uint32_t span, std::uint32_t skip, std::uint32_t x_size,
    std::uint32_t y_size, half *py) {
  const std::uint32_t i = IDX;
  if (i < y_size) py[(i / span) * skip + (i % span)] = px[i % x_size];
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::concat_fw_impl(
    const std::vector<const Tensor *> &xs, std::uint32_t dim, Tensor &y) {
  const std::uint32_t new_bs = y.shape().batch();
  const std::uint32_t base = y.shape().lower_volume(dim);
  const std::uint32_t skip = base * y.shape()[dim];
  std::uint32_t repeat = y.shape().volume() / skip;
  CUDA_CALL(::hipSetDevice(dev_id_));
  std::uint32_t offset = 0;
  for (const Tensor *x : xs) {
    const std::uint32_t span = base * x->shape()[dim];
    const std::uint32_t x_size = span * repeat * x->shape().batch();
    const std::uint32_t y_size = span * repeat * new_bs;
    const std::uint32_t num_blocks = GRID_SIZE(y_size, dim1_x_);
    ::concat_fw_dev<<<num_blocks, dim1_x_>>>(
       CDATA(half, *x), span, skip, x_size, y_size, MDATA(half, y) + offset);
    offset += span;
  }
}

}  // namespace devices
}  // namespace primitiv
