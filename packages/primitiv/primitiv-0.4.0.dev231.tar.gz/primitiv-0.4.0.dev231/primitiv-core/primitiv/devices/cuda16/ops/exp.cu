#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

CUDA16_KERNEL_FW_X(exp, ::expf(X_VAL));
CUDA16_KERNEL_BW_X(exp, Y_VAL * GY_VAL);

}  // namespace

namespace primitiv {
namespace devices {

CUDA16_DEV_FW_X(exp);
CUDA16_DEV_BW_X(exp);

}  // namespace devices
}  // namespace primitiv
