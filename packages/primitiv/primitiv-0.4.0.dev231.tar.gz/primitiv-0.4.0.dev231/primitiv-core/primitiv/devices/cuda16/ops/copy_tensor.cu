#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace primitiv {
namespace devices {

void CUDA16::copy_tensor_impl(const Tensor &x, Tensor &y) {
  switch (x.device().type()) {
    case DeviceType::NAIVE:
      reset_tensor_by_array(CDATA(float, x), y);
      break;
    //case DeviceType::CUDA:
      // TODO(odashi): Implement this section.
    case DeviceType::CUDA16:
      CUDA_CALL(::hipSetDevice(dev_id_));
      // NOTE(odashi):
      // If source/destination devices use the unified memory space on the 64
      // bits machine, we can perform ::hipMemcpy to copy data beyond devices.
      CUDA_CALL(::hipMemcpyAsync(
            MDATA(half, y), CDATA(half, x),
            sizeof(half) * x.shape().size(),
            hipMemcpyDeviceToDevice, 0));
      break;
    default:
      reset_tensor_by_vector(x.to_vector(), y);
  }
}

}  // namespace devices
}  // namespace primitiv
