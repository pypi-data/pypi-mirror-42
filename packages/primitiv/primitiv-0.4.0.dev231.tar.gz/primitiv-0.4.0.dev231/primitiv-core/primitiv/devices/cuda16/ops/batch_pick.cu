#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void batch_pick_fw_dev(
    const half *px, const std::uint32_t *pi,
    std::uint32_t si, std::uint32_t sy, half *py) {
  const std::uint32_t t = IDX;
  const std::uint32_t ox = pi[blockIdx.y * si] * sy;
  const std::uint32_t oy = blockIdx.y * sy;
  if (t < sy) py[oy + t] = px[ox + t];
}

DECL_ATOMIC_OP(atomicHAdd, ::__fadd_rn);

__global__ void batch_pick_bw_dev(
    const half *pgy, const std::uint32_t *pi,
    std::uint32_t si, std::uint32_t sy, half *pgx) {
  const std::uint32_t t = IDX;
  const std::uint32_t ox = pi[blockIdx.y * si] * sy;
  const std::uint32_t oy = blockIdx.y * sy;
  if (t < sy) ::atomicHAdd(pgx, ox + t, ::__half2float(pgy[oy + t]));
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::batch_pick_fw_impl(
    const Tensor &x, const std::vector<std::uint32_t> &ids, Tensor &y) {
  const std::uint32_t sy = y.shape().volume();
  const std::uint32_t g1 = GRID_SIZE(sy, dim1_x_);
  const std::uint32_t bs = y.shape().batch();

  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        ids_ptr_.get(), ids.data(), sizeof(std::uint32_t) * ids.size(),
        hipMemcpyHostToDevice));
  ::batch_pick_fw_dev<<<dim3(g1, bs), dim1_x_>>>(
      CDATA(half, x), static_cast<const std::uint32_t *>(ids_ptr_.get()),
      ids.size() > 1, sy, MDATA(half, y));
}

void CUDA16::batch_pick_bw_impl(
    const Tensor &gy, const std::vector<std::uint32_t>& ids, Tensor &gx) {
  const std::uint32_t sy = gy.shape().volume();
  const std::uint32_t g1 = GRID_SIZE(sy, dim1_x_);
  const std::uint32_t bs = gy.shape().batch();

  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        ids_ptr_.get(), ids.data(), sizeof(std::uint32_t) * ids.size(),
        hipMemcpyHostToDevice));
  ::batch_pick_bw_dev<<<dim3(g1, bs), dim1_x_>>>(
      CDATA(half, gy), static_cast<const std::uint32_t *>(ids_ptr_.get()),
      ids.size() > 1, sy, MDATA(half, gx));
}

}  // namespace devices
}  // namespace primitiv
