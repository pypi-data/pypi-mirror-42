#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <cstring>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace primitiv {
namespace devices {

void CUDA16::conv2d_fw_impl(
    const Tensor &x, const Tensor &w,
    std::uint32_t padding0, std::uint32_t padding1,
    std::uint32_t stride0, std::uint32_t stride1,
    std::uint32_t dilation0, std::uint32_t dilation1,
    Tensor &y) {

#ifdef PRIMITIV_USE_CUDNN

  const Shape x_shape = x.shape();
  const Shape w_shape = w.shape();
  const Shape y_shape = y.shape();

  // Specifies a target device.
  CUDA_CALL(::hipSetDevice(dev_id_));

  // Prepares descriptors.
  const cuda::CuDNNTensorDescriptor x_desc(
      w_shape.has_batch() ? 1 : x_shape.batch(),
      x_shape[2], x_shape[1], x_shape[0],
      ::HIPDNN_DATA_HALF);
  const cuda::CuDNNTensorDescriptor y_desc(
      w_shape.has_batch() ? 1 : y_shape.batch(),
      y_shape[2], y_shape[1], y_shape[0],
      ::HIPDNN_DATA_HALF);
  const cuda::CuDNNFilterDescriptor w_desc(
      w_shape[3], w_shape[2], w_shape[1], w_shape[0],
      ::HIPDNN_DATA_HALF);
  const cuda::CuDNNConvolutionDescriptor conv_desc(
      padding1, padding0, stride1, stride0, dilation1, dilation0,
      ::HIPDNN_DATA_HALF);

  // Obtains the most efficient algorithm.
  ::hipdnnConvolutionFwdAlgo_t algo;
  CUDNN_CALL(::hipdnnGetConvolutionForwardAlgorithm(
        state_->cudnn.get(),
        x_desc.get(), w_desc.get(), conv_desc.get(), y_desc.get(),
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

  // Obtains workspace size/memory.
  std::size_t ws_size;
  CUDNN_CALL(::hipdnnGetConvolutionForwardWorkspaceSize(
        state_->cudnn.get(),
        x_desc.get(), w_desc.get(), conv_desc.get(), y_desc.get(),
        algo, &ws_size));
  std::shared_ptr<void> ws_ptr = state_->pool.allocate(ws_size);

  // Performs forward operations.
  const std::size_t x_shift = x_shape.has_batch() * x_shape.volume();
  const std::size_t w_shift = w_shape.volume();
  const std::size_t y_shift = y_shape.volume();
  const float alpha = 1.f;
  const float beta = 0.f;
  const half *x_ptr = CDATA(half, x);
  const half *w_ptr = CDATA(half, w);
  half *y_ptr = MDATA(half, y);
  for (std::uint32_t bn = 0; bn < w_shape.batch(); ++bn) {
    CUDNN_CALL(::hipdnnConvolutionForward(
          state_->cudnn.get(),
          &alpha, x_desc.get(), x_ptr, w_desc.get(), w_ptr,
          conv_desc.get(), algo, ws_ptr.get(), ws_size,
          &beta, y_desc.get(), y_ptr));
    x_ptr += x_shift;
    w_ptr += w_shift;
    y_ptr += y_shift;
  }

#else  // PRIMITIV_USE_CUDNN

  static_cast<void>(x);
  static_cast<void>(w);
  static_cast<void>(padding0);
  static_cast<void>(padding1);
  static_cast<void>(stride0);
  static_cast<void>(stride1);
  static_cast<void>(dilation0);
  static_cast<void>(dilation1);
  static_cast<void>(y);
  PRIMITIV_THROW_NOT_IMPLEMENTED;

#endif  // PRIMITIV_USE_CUDNN
}

void CUDA16::conv2d_bw_impl(
    const Tensor &x, const Tensor &w, const Tensor &, const Tensor &gy,
    std::uint32_t padding0, std::uint32_t padding1,
    std::uint32_t stride0, std::uint32_t stride1,
    std::uint32_t dilation0, std::uint32_t dilation1,
    Tensor &gx, Tensor &gw) {

#ifdef PRIMITIV_USE_CUDNN

  const Shape x_shape = x.shape();
  const Shape w_shape = w.shape();
  const Shape y_shape = gy.shape();

  // Specifies a target device.
  CUDA_CALL(::hipSetDevice(dev_id_));

  // Prepares descriptors.
  const cuda::CuDNNTensorDescriptor x_desc(
      w_shape.has_batch() ? 1 : x_shape.batch(),
      x_shape[2], x_shape[1], x_shape[0],
      ::HIPDNN_DATA_HALF);
  const cuda::CuDNNTensorDescriptor y_desc(
      w_shape.has_batch() ? 1 : y_shape.batch(),
      y_shape[2], y_shape[1], y_shape[0],
      ::HIPDNN_DATA_HALF);
  const cuda::CuDNNFilterDescriptor w_desc(
      w_shape[3], w_shape[2], w_shape[1], w_shape[0],
      ::HIPDNN_DATA_HALF);
  const cuda::CuDNNConvolutionDescriptor conv_desc(
      padding1, padding0, stride1, stride0, dilation1, dilation0,
      ::HIPDNN_DATA_HALF);

  // Obtains the most efficient algorithms.
  ::hipdnnConvolutionBwdDataAlgo_t x_algo;
  ::hipdnnConvolutionBwdFilterAlgo_t w_algo;
  CUDNN_CALL(::hipdnnGetConvolutionBackwardDataAlgorithm(
        state_->cudnn.get(),
        w_desc.get(), y_desc.get(), conv_desc.get(), x_desc.get(),
        HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &x_algo));
  CUDNN_CALL(::hipdnnGetConvolutionBackwardFilterAlgorithm(
        state_->cudnn.get(),
        x_desc.get(), y_desc.get(), conv_desc.get(), w_desc.get(),
        HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &w_algo));

  // Obtains workspace sizes/memory.
  std::size_t x_ws_size, w_ws_size;
  CUDNN_CALL(::hipdnnGetConvolutionBackwardDataWorkspaceSize(
        state_->cudnn.get(),
        w_desc.get(), y_desc.get(), conv_desc.get(), x_desc.get(),
        x_algo, &x_ws_size));
  CUDNN_CALL(::hipdnnGetConvolutionBackwardFilterWorkspaceSize(
        state_->cudnn.get(),
        x_desc.get(), y_desc.get(), conv_desc.get(), w_desc.get(),
        w_algo, &w_ws_size));
  const std::size_t ws_size = std::max(x_ws_size, w_ws_size);
  std::shared_ptr<void> ws_ptr = state_->pool.allocate(ws_size);

  // Performs backward operations.
  const std::size_t x_shift = x_shape.has_batch() * x_shape.volume();
  const std::size_t w_shift = w_shape.volume();
  const std::size_t y_shift = y_shape.volume();
  const float alpha = 1.f;
  const float beta = 1.f;
  const half *x_ptr = CDATA(half, x);
  const half *w_ptr = CDATA(half, w);
  const half *gy_ptr = CDATA(half, gy);
  half *gx_ptr = MDATA(half, gx);
  half *gw_ptr = MDATA(half, gw);
  for (std::uint32_t bn = 0; bn < w_shape.batch(); ++bn) {
    CUDNN_CALL(::hipdnnConvolutionBackwardData(
          state_->cudnn.get(),
          &alpha, w_desc.get(), w_ptr, y_desc.get(), gy_ptr,
          conv_desc.get(), x_algo, ws_ptr.get(), ws_size,
          &beta, x_desc.get(), gx_ptr));
    CUDNN_CALL(::hipdnnConvolutionBackwardFilter(
          state_->cudnn.get(),
          &alpha, x_desc.get(), x_ptr, y_desc.get(), gy_ptr,
          conv_desc.get(), w_algo, ws_ptr.get(), ws_size,
          &beta, w_desc.get(), gw_ptr));
    x_ptr += x_shift;
    w_ptr += w_shift;
    gy_ptr += y_shift;
    gx_ptr += x_shift;
    gw_ptr += w_shift;
  }

#else  // PRIMITIV_USE_CUDNN

  static_cast<void>(x);
  static_cast<void>(w);
  static_cast<void>(gy);
  static_cast<void>(padding0);
  static_cast<void>(padding1);
  static_cast<void>(stride0);
  static_cast<void>(stride1);
  static_cast<void>(dilation0);
  static_cast<void>(dilation1);
  static_cast<void>(gx);
  static_cast<void>(gw);
  PRIMITIV_THROW_NOT_IMPLEMENTED;

#endif  // PRIMITIV_USE_CUDNN
}

}  // namespace devices
}  // namespace primitiv
