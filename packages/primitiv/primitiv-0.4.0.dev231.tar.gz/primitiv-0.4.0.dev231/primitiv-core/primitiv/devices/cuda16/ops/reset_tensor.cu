#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

__global__ void set_const_dev(float k, std::uint32_t size, half *py) {
  const std::uint32_t i = IDX;
  if (i < size) py[i] = ::__float2half(k);
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::reset_tensor_impl(float k, Tensor &x) {
  const std::uint32_t size = x.shape().size();
  const std::uint32_t num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::set_const_dev<<<num_blocks, dim1_x_>>>(k, size, MDATA(half, x));
}

}  // namespace devices
}  // namespace primitiv
