#include "hip/hip_runtime.h"
#include <primitiv/config.h>

#include <cstring>

#include <primitiv/devices/cuda16/device.h>
#include <primitiv/devices/cuda16/ops/common.h>
#include <primitiv/internal/cuda/utils.h>

namespace {

#if CUDART_VERSION >= 9000
__global__ void set_gemm_ptrs(
    const half *pa, const half *pb, const half *py,
    std::uint32_t na, std::uint32_t nb, std::uint32_t ny, std::uint32_t bs,
    const half **ptrs) {
  const std::uint32_t i = IDX;
  if (i < bs) {
    ptrs[i] = pa + i * na;
    ptrs[i + bs] = pb + i * nb;
    ptrs[i + 2 * bs] = py + i * ny;
  }
}
#endif  // CUDART_VERSION

inline half half_zero() {
  static_assert(sizeof(half) == sizeof(std::uint16_t), "");
  constexpr std::uint16_t repr = 0x0000;
  half ret;
  std::memcpy(&ret, &repr, sizeof(half));
  return ret;
}

inline half half_one() {
  static_assert(sizeof(half) == sizeof(std::uint16_t), "");
  constexpr std::uint16_t repr = 0x3c00;
  half ret;
  std::memcpy(&ret, &repr, sizeof(half));
  return ret;
}

}  // namespace

namespace primitiv {
namespace devices {

void CUDA16::matmul_fw_impl(const Tensor &a, const Tensor &b, Tensor &y) {
  const std::uint32_t di = a.shape()[0];
  const std::uint32_t dj = a.shape()[1];
  const std::uint32_t dk = b.shape()[1];
  constexpr float f_alpha = 1.;
  constexpr float f_beta = 0.;
  const half h_alpha = ::half_one();
  const half h_beta = ::half_zero();

  CUDA_CALL(::hipSetDevice(dev_id_));

  if (a.shape().has_batch()) {
    // Do gemm multiple times.
    const half *pa = CDATA(half, a);
    const half *pb = CDATA(half, b);
    half *py = MDATA(half, y);
    const std::uint32_t na = di * dj;
    const std::uint32_t nb = b.shape().has_batch() * dj * dk;
    const std::uint32_t ny = di * dk;
    const std::uint32_t bs = a.shape().batch();

    if (support_half_ops_) {

#if CUDART_VERSION >= 9000

      std::shared_ptr<void> ptrs = state_->pool.allocate(
          3 * bs * sizeof(void *));
      const half **fptrs = static_cast<const half **>(ptrs.get());

      const std::uint32_t gs = GRID_SIZE(bs, dim1_x_);

      ::set_gemm_ptrs<<<gs, dim1_x_>>>(pa, pb, py, na, nb, ny, bs, fptrs);
      CUBLAS_CALL(::hipblasHgemmBatched(
            state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
            di, dk, dj,
            &h_alpha, fptrs, di, fptrs + bs, dj,
            &h_beta, const_cast<half **>(fptrs) + 2 * bs, di,
            bs));

#else  // CUDART_VERSION < 9000

      for (std::uint32_t n = 0; n < bs; ++n) {
        CUBLAS_CALL(::hipblasHgemm(
              state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
              di, dk, dj,
              &h_alpha, pa + n * na, di, pb + n * nb, dj,
              &h_beta, py + n * ny, di));
      }

#endif  // CUDART_VERSION

    } else {
      for (std::uint32_t n = 0; n < bs; ++n) {
        CUBLAS_CALL(::cublasSgemmEx(
              state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
              di, dk, dj,
              &f_alpha,
              pa + n * na, HIP_R_16F,
              di, pb + n * nb, HIP_R_16F, dj,
              &f_beta,
              py + n * ny, HIP_R_16F, di));
      }
    }

  } else {
    // Do gemm only once to calculate the product with a combined matrix.
    if (support_half_ops_) {
      CUBLAS_CALL(::hipblasHgemm(
            state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
            di, dk * b.shape().batch(), dj,
            &h_alpha, CDATA(half, a), di, CDATA(half, b), dj,
            &h_beta, MDATA(half, y), di));
    } else {
      CUBLAS_CALL(::cublasSgemmEx(
            state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
            di, dk * b.shape().batch(), dj,
            &f_alpha,
            CDATA(half, a), HIP_R_16F, di,
            CDATA(half, b), HIP_R_16F, dj,
            &f_beta,
            MDATA(half, y), HIP_R_16F, di));
    }
  }
}

void CUDA16::matmul_bw_impl(
    const Tensor &a, const Tensor &b, const Tensor &, const Tensor &gy,
    Tensor &ga, Tensor &gb) {
  // ga += gy . b^T
  // gb += a^T . gy
  const std::uint32_t di = a.shape()[0];
  const std::uint32_t dj = a.shape()[1];
  const std::uint32_t dk = b.shape()[1];
  const float f_alpha = 1.;
  const float f_beta = 1.;
  const half h_alpha = ::half_one();
  const half h_beta = ::half_one();

  CUDA_CALL(::hipSetDevice(dev_id_));

  if (a.shape().has_batch()) {
    // Do gemm multiple times.
    const half *pa = CDATA(half, a);
    const half *pb = CDATA(half, b);
    const half *pgy = CDATA(half, gy);
    half *pga = MDATA(half, ga);
    half *pgb = MDATA(half, gb);
    const std::uint32_t na = di * dj;
    const std::uint32_t nb = b.shape().has_batch() * dj * dk;
    const std::uint32_t ny = di * dk;
    const std::uint32_t bs = a.shape().batch();

    if (support_half_ops_) {

#if CUDART_VERSION >= 9000

      std::shared_ptr<void> ptrs = state_->pool.allocate(3 * bs * sizeof(void *));
      const half **fptrs = static_cast<const half **>(ptrs.get());

      const std::uint32_t gs = GRID_SIZE(bs, dim1_x_);

      ::set_gemm_ptrs<<<gs, dim1_x_>>>(pgy, pb, pga, ny, nb, na, bs, fptrs);
      CUBLAS_CALL(::hipblasHgemmBatched(
            state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
            di, dj, dk,
            &h_alpha, fptrs, di, fptrs + bs, dj,
            &h_beta, const_cast<half **>(fptrs) + 2 * bs, di,
            bs));

      if (nb > 0 /* `b` has minibatch */) {
        ::set_gemm_ptrs<<<gs, dim1_x_>>>(pa, pgy, pgb, na, ny, nb, bs, fptrs);
        CUBLAS_CALL(::hipblasHgemmBatched(
              state_->cublas.get(), ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
              dj, dk, di,
              &h_alpha, fptrs, di, fptrs + bs, di,
              &h_beta, const_cast<half **>(fptrs) + 2 * bs, dj,
              bs));
      } else {
        // NOTE(odashi):
        // `hipblasHgemmBatched` can not be used due to a data race against
        // shared values in `b` by multiple GEMM operations.
        for (std::uint32_t batch = 0; batch < bs; ++batch) {
          CUBLAS_CALL(::hipblasHgemm(
                state_->cublas.get(), ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
                dj, dk, di,
                &h_alpha, pa + batch * na, di, pgy + batch * ny, di,
                &h_beta, pgb, dj));
        }
      }

#else  // CUDART_VERSION < 9000

      for (std::uint32_t n = 0; n < bs; ++n) {
        CUBLAS_CALL(::hipblasHgemm(
              state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
              di, dj, dk,
              &h_alpha, pgy + n * ny, di, pb + n * nb, dj,
              &h_beta, pga + n * na, di));
        CUBLAS_CALL(::hipblasHgemm(
              state_->cublas.get(), ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
              dj, dk, di,
              &h_alpha, pa + n * na, di, pgy + n * ny, di,
              &h_beta, pgb + n * nb, dj));
      }

#endif  // CURART_VERSION

    } else {
      for (std::uint32_t n = 0; n < bs; ++n) {
        CUBLAS_CALL(::cublasSgemmEx(
              state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
              di, dj, dk,
              &f_alpha,
              pgy + n * ny, HIP_R_16F, di,
              pb + n * nb, HIP_R_16F, dj,
              &f_beta,
              pga + n * na, HIP_R_16F, di));
        CUBLAS_CALL(::cublasSgemmEx(
              state_->cublas.get(), ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
              dj, dk, di,
              &f_alpha,
              pa + n * na, HIP_R_16F, di,
              pgy + n * ny, HIP_R_16F, di,
              &f_beta,
              pgb + n * nb, HIP_R_16F, dj));
      }
    }

  } else {
    // Do gemm only once to calculate the product with a combined matrix.
    if (support_half_ops_) {
      CUBLAS_CALL(::hipblasHgemm(
            state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
            di, dj, dk * b.shape().batch(),
            &h_alpha, CDATA(half, gy), di, CDATA(half, b), dj,
            &h_beta, MDATA(half, ga), di));
      CUBLAS_CALL(::hipblasHgemm(
            state_->cublas.get(), ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
            dj, dk * b.shape().batch(), di,
            &h_alpha, CDATA(half, a), di, CDATA(half, gy), di,
            &h_beta, MDATA(half, gb), dj));
    } else {
      CUBLAS_CALL(::cublasSgemmEx(
            state_->cublas.get(), ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
            di, dj, dk * b.shape().batch(),
            &f_alpha,
            CDATA(half, gy), HIP_R_16F, di,
            CDATA(half, b), HIP_R_16F, dj,
            &f_beta,
            MDATA(half, ga), HIP_R_16F, di));
      CUBLAS_CALL(::cublasSgemmEx(
            state_->cublas.get(), ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
            dj, dk * b.shape().batch(), di,
            &f_alpha,
            CDATA(half, a), HIP_R_16F, di,
            CDATA(half, gy), HIP_R_16F, di,
            &f_beta,
            MDATA(half, gb), HIP_R_16F, dj));
    }
  }
}

}  // namespace devices
}  // namespace primitiv
